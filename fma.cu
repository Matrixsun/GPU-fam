#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * Demonstration of inline PTX (assembly language) usage in CUDA kernels
 */

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#define STREAM 4
#define INTER 1000000
__global__ void fma_fp16(int a, int b, int c, int* res)
{
	int a1=a;
	int b1=b;
	int c1=c;

	int a2=a+res[0];
	int b2=b+res[0];
	int c2=c+res[0];
	int a3=a+res[1];
	int b3=b+res[1];
	int c3=c+res[1];
	int a4=a+res[2];
	int b4=b+res[2];
	int c4=c+res[2];

	int elemID = blockIdx.x * blockDim.x + threadIdx.x;

	int i;
	for(i=0;i<INTER;i++)
	{
		asm volatile ("fma.rn.f16x2 %0, %1, %2, %3;" : "=r"(c1) : "r"(a1), "r"(b1), "r"(c1));
		asm volatile ("fma.rn.f16x2 %0, %1, %2, %3;" : "=r"(c2) : "r"(a2), "r"(b2), "r"(c2));
		asm volatile ("fma.rn.f16x2 %0, %1, %2, %3;" : "=r"(c3) : "r"(a3), "r"(b3), "r"(c3));
		asm volatile ("fma.rn.f16x2 %0, %1, %2, %3;" : "=r"(c4) : "r"(a4), "r"(b4), "r"(c4));
	}
	res[STREAM*elemID]=c1;
	res[STREAM*elemID+1]=c2;
	res[STREAM*elemID+2]=c3;
	res[STREAM*elemID+3]=c4;
}

__global__ void fma_fp32(int a, int b, int c, int* res)
{
	int a1=a;
	int b1=b;
	int c1=c;

	int a2=a+res[0];
	int b2=b+res[0];
	int c2=c+res[0];
	int a3=a+res[1];
	int b3=b+res[1];
	int c3=c+res[1];
	int a4=a+res[2];
	int b4=b+res[2];
	int c4=c+res[2];

	int elemID = blockIdx.x * blockDim.x + threadIdx.x;

	int i;
	for(i=0;i<INTER;i++)
	{
		asm volatile ("fma.rn.f32 %0, %1, %2, %3;" : "=r"(c1) : "r"(a1), "r"(b1), "r"(c1));
		asm volatile ("fma.rn.f32 %0, %1, %2, %3;" : "=r"(c2) : "r"(a2), "r"(b2), "r"(c2));
		asm volatile ("fma.rn.f32 %0, %1, %2, %3;" : "=r"(c3) : "r"(a3), "r"(b3), "r"(c3));
		asm volatile ("fma.rn.f32 %0, %1, %2, %3;" : "=r"(c4) : "r"(a4), "r"(b4), "r"(c4));
	}
	res[STREAM*elemID]=c1;
	res[STREAM*elemID+1]=c2;
	res[STREAM*elemID+2]=c3;
	res[STREAM*elemID+3]=c4;
}
 
__global__ void fma_int8(int a, int b, int c, int* res)
{
	int a1=a;
	int b1=b;
	int c1=c;
	int a2=a+res[0];
	int b2=b+res[0];
	int c2=c+res[0];
	int a3=a+res[1];
	int b3=b+res[1];
	int c3=c+res[1];
	int a4=a+res[2];
	int b4=b+res[2];
	int c4=c+res[2];

	int elemID = blockIdx.x * blockDim.x + threadIdx.x;

	int i;
	for(i=0;i<INTER;i++)
	{
		asm volatile ("dp4a.s32.s32 %0, %1, %2, %3;" : "=r"(c1) : "r"(a1), "r"(b1), "r"(c1));
		asm volatile ("dp4a.s32.s32 %0, %1, %2, %3;" : "=r"(c2) : "r"(a2), "r"(b2), "r"(c2));
		asm volatile ("dp4a.s32.s32 %0, %1, %2, %3;" : "=r"(c3) : "r"(a3), "r"(b3), "r"(c3));
		asm volatile ("dp4a.s32.s32 %0, %1, %2, %3;" : "=r"(c4) : "r"(a4), "r"(b4), "r"(c4));
	}
	res[STREAM*elemID]=c1;
	res[STREAM*elemID+1]=c2;
	res[STREAM*elemID+2]=c3;
	res[STREAM*elemID+3]=c4;
}


int main(int argc, char **argv)
{
	printf("Theoretical computing power tests\n");
	int threadNum=512;
	int blockNum=400;
	const int N = STREAM*blockNum*threadNum;

        hipEvent_t start, stop;
	int dev = findCudaDevice(argc, (const char **) argv);

	if (dev == -1)
	{
		return EXIT_FAILURE;
	}
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));


	int *d_ptr;
	checkCudaErrors(hipMalloc(&d_ptr, N * sizeof(int)));

	int *h_ptr;
	checkCudaErrors(hipHostMalloc(&h_ptr, N * sizeof(int)));

	dim3 cudaBlockSize(threadNum,1,1);
	dim3 cudaGridSize(blockNum, 1, 1);

	checkCudaErrors(hipEventRecord(start, NULL));
	int round=1;
	int i;
	for(i=0;i<round;i++)
	{
		fma_fp16<<<cudaGridSize, cudaBlockSize>>>(3,4,5,d_ptr);
	} 
	checkCudaErrors(hipEventRecord(stop, NULL));
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());


	checkCudaErrors(hipMemcpy(h_ptr, d_ptr, N *sizeof(int), hipMemcpyDeviceToHost));
	printf("FP16 done \n");
	float msecTotal = 0.0f;
	double ops=(double)STREAM*(double)INTER*(double)threadNum*(double)blockNum*4*round;
	checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));
	double opsps=(ops*1.0e-9f)/(msecTotal/1000.0f);
	printf(
			"Performance= %.2fG FP16 op/s, Time= %.3f msec, Size= %.0f Ops\n",
			opsps,
			msecTotal,
			ops);

	checkCudaErrors(hipEventRecord(start, NULL));
	for(i=0;i<round;i++)
	{
		fma_int8<<<cudaGridSize, cudaBlockSize>>>(3,4,5,d_ptr);
	} 
	checkCudaErrors(hipEventRecord(stop, NULL));
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());


	checkCudaErrors(hipMemcpy(h_ptr, d_ptr, N *sizeof(int), hipMemcpyDeviceToHost));
	printf("int8 done \n");
	ops=(double)STREAM*(double)INTER*(double)threadNum*(double)blockNum*8*round;
	checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));
	opsps=(ops*1.0e-9f)/(msecTotal/1000.0f);
	printf(
			"Performance= %.2fG INT8 op/s, Time= %.3f msec, Size= %.0f Ops\n",
			opsps,
			msecTotal,
			ops);

	checkCudaErrors(hipEventRecord(start, NULL));
	for(i=0;i<round;i++)
	{
		fma_fp32<<<cudaGridSize, cudaBlockSize>>>(3,4,5,d_ptr);
	} 
	checkCudaErrors(hipEventRecord(stop, NULL));
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());


	checkCudaErrors(hipMemcpy(h_ptr, d_ptr, N *sizeof(int), hipMemcpyDeviceToHost));
	printf("fp32 done \n");
	ops=(double)STREAM*(double)INTER*(double)threadNum*(double)blockNum*2*round;
	checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));
	opsps=(ops*1.0e-9f)/(msecTotal/1000.0f);
	printf(
			"Performance= %.2fG FP32 op/s, Time= %.3f msec, Size= %.0f Ops\n",
			opsps,
			msecTotal,
			ops);



	checkCudaErrors(hipFree(d_ptr));
	checkCudaErrors(hipHostFree(h_ptr));


	// Calling hipProfilerStop causes all profile data to be
	// flushed before the application exits
	checkCudaErrors(hipProfilerStop());

	return 0;
}
